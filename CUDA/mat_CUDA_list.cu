
#include <hip/hip_runtime.h>
#include <iostream>
#include <cmath>
#include <chrono>
#include <fstream>
#include <vector>

using namespace std;

// Matrix size (assuming square matrices)
int N;

// CUDA kernel for matrix multiplication
__global__ void matrixMultiplication(float* A, float* B, float* C, int n){

	int row = blockIdx.y * blockDim.y + threadIdx.y;
	int col = blockIdx.x * blockDim.x + threadIdx.x;

	if (row < n && col < n) {
		float sum = 0.0f;
		for (int i = 0; i < n; ++i) {
			sum += A[row * n + i] * B[i * n + col];
		}
		C[row * n + col] = sum;
	}
}

int main() {
	// Storing data into file so that we can use python to plot the graphs
    	string filename = "mat_cuda_sub_2.out";
    	ofstream outFile;
    	outFile.open(filename);

    	if(outFile.fail()){
        	cout<<"Error Opening the file"<<endl;
    	}

	
	//Matrix will be checked for sizes 1-4096
        int elements = 1500;

   	vector<double> time;
    	vector<int> dimension;
    	
    	for(int i=1;i<elements;i+=5)
        	dimension.push_back(i);

    	for(int i=0;i<dimension.size();i++){

		// Matrix allocation and initialization
		N=dimension[i];
        	float *h_A, *h_B, *h_C;
		size_t matrix_size = N * N * sizeof(float);

		h_A = new float[N * N];
		h_B = new float[N * N];
		h_C = new float[N * N];

		for (int i = 0; i < N * N; ++i) {
			h_A[i] = 1;
			h_B[i] = 2;
			h_C[i] = 0;
		}
	
		// start the timer for the execution phase
		auto start = chrono::high_resolution_clock::now();

		// CUDA memory allocation
		float *d_A, *d_B, *d_C;
		hipMalloc(&d_A, matrix_size);
		hipMalloc(&d_B, matrix_size);
		hipMalloc(&d_C, matrix_size);

		// Copy matrices from host to device
		hipMemcpy(d_A, h_A, matrix_size, hipMemcpyHostToDevice);
		hipMemcpy(d_B, h_B, matrix_size, hipMemcpyHostToDevice);

		// Define block and grid dimensions
		dim3 blockSize(16, 16);
		dim3 gridSize((N + blockSize.x - 1) / blockSize.x, (N + blockSize.y - 1) / blockSize.y);

		// Launch kernel
		matrixMultiplication<<<gridSize, blockSize>>>(d_A, d_B, d_C, N);

		// Copy result from device to host
		hipMemcpy(h_C, d_C, matrix_size, hipMemcpyDeviceToHost);

		//Free CUDA memory
		hipFree(d_A);
		hipFree(d_B);
		hipFree(d_C);

		// Free host memory
		delete[] h_A;
		delete[] h_B;
		delete[] h_C;
   
		auto end = chrono::high_resolution_clock::now();
	   	chrono::duration<double> duration = end - start;
		
		cout<<N<<" "<<N<<endl;
    		cout<<"Parallel Execution Time :  "<< duration.count() << " seconds\n";
		time.push_back(duration.count());
	}
	for(auto element : time)
	        outFile<<element*1000<<endl;

	return 0;
}
